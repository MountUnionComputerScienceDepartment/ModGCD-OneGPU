#include "hip/hip_runtime.h"
/*  GmpCudaDevice.cu -- provides constructor for GmpCudaDevice objects.

  Implementation of the modular integer gcd algorithm using L <= 32 bit moduli.
  
  Reference: Weber, Trevisan, Martins 2005. A Modular Integer GCD algorithm
             Journal of Algorithms 54, 2 (February, 2005) 152-167.

             Note that there is an error in Fig. 2, which shows that the
             final result can be recovered as the mixed radix representation
             is calculated.  In actuality, all the mixed radix digits and moduli
             must be computed before the actual GCD can be recovered.

  This version is for a single device.

  Runs in CUDA 9.
  
  Based on initial work by
  Authors:  Justin Brew, Anthony Rizzo, Kenneth Weber
            Mount Union College
            June 25, 2009

  Further revisions by 
  K. Weber  University of Mount Union
            weberk@mountunion.edu
            
  History:  Basic 16 bit version      January, 2010 
  
            Additional modifications  July, 2010
            
            Further simplifications   February, 2011
            including using float 
            operations for modulus
            
            Reorganized               March 8, 2011
            
            Eliminated parallel       June 22, 2011
            conversion to standard
            representation
                                  
            "Final" cleanup           July, 2011

            Modified to allow up      June, 2012
	          to 32 bit moduli         

            Made object-oriented      August, 2012

            More cleanup              January, 2013
            limited to arch >= 2.0
            (anyPair uses __ballot)

            Bug fixed in barrier      March, 2014
						Uses fixed number of
					  threads, but arbitrary
            number of moduli.
            Also overlaps communi-
            cation with computation.

            Further cleanup           July, 2014
            
            Ported to CUDA 9.         January 11, 2018
            Uses shuffle mechanism
            for the min operation
            and ballot mechanism
            to select a nonzero
            value

            Put GmpCudaDevice::gcd    January 22, 2018
            in its own file named
            GmpCudaDevice-gcd.cu
            Added capability to use 
            more than warpSize SMs.
            
            Split out GmpCudaBarrier  January, 2018
            in files GmpCudaBarrier.h
            and GmpCudaBarrier.cu

            Modified to allow large   February 17, 2018
            grid sizes up to maximum 
            occupancy.  
            
            Corrected errors in       May 2, 2018
            modInv.
*/

//  Enforce use of CUDA 9 at compile time.
#if defined(CUDART_VERSION) && CUDART_VERSION >= 9000
#else
#error Requires CUDA 9 or more recent
#endif

#include <cassert>
#include <hip/hip_runtime.h>
#include "GmpCuda.h"
#if defined(USE_COOP_GROUPS)
#include <iostream>
#endif
using namespace GmpCuda;

//  Initialize the CUDA device.  The device to use can be set by hipSetDevice.
//  If 0 < n < the device's number of SMs,
//  the device's number of SMs is changed to n.
//  Also initializes the global barrier.
GmpCudaDevice::GmpCudaDevice(int n)
{
  static GcdKernelPtr_t gcdKernelPtr = getGcdKernelPtr();

  assert(hipSuccess == hipGetDevice(&deviceNum));

  //  Initialize the device properties values.
  struct hipDeviceProp_t props;
  assert(hipSuccess == hipGetDeviceProperties(&props, deviceNum));

  assert(props.warpSize == WARP_SZ);  //  Assume a fixed warp size of 32 for the forseeable future.
  
  assert(BLOCK_SZ <= props.maxThreadsPerBlock);
  
#if defined(USE_COOP_GROUPS)
  deviceSupportsCooperativeLaunch = (props.cooperativeLaunch == 1);
  std::cerr << "Using cooperative groups if device supports it." << std::endl;
#endif

  //  Limit the grid, and thus, the barrier size.
  int gcdOccupancy;
  assert(hipSuccess == hipOccupancyMaxActiveBlocksPerMultiprocessor(&gcdOccupancy, gcdKernelPtr, BLOCK_SZ, 0));
  maxGridSize = min(BLOCK_SZ, props.multiProcessorCount * gcdOccupancy);    
  if (0 < n && n < maxGridSize)
    maxGridSize = n;
    
  barrier = new GmpCudaBarrier(maxGridSize);
  
    //  Copy moduli to device.
  size_t maxModuli = maxGridSize * BLOCK_SZ;
  assert(hipSuccess == hipMalloc(&moduliList, maxModuli * sizeof(uint32_t)));
  assert(hipSuccess == hipMemcpy(moduliList, moduli, maxModuli * sizeof(uint32_t), hipMemcpyHostToDevice));
}

GmpCudaDevice::~GmpCudaDevice()
{
  assert(hipSuccess == hipFree(moduliList));
  delete barrier;
}
