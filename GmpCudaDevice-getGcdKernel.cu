#include "hip/hip_runtime.h"
/*  GmpCudaDevice-getGcdKernel.cu -- provides GmpCudaDevice::getGcdKernel method
                                     (includes the gcd kernel code).

  Implementation of the modular integer gcd algorithm using L <= 32 bit moduli.
  
  Reference: Weber, Trevisan, Martins 2005. A Modular Integer GCD algorithm
             Journal of Algorithms 54, 2 (February, 2005) 152-167.

             Note that there is an error in Fig. 2, which shows that the
             final result can be recovered as the mixed radix representation
             is calculated.  In actuality, all the mixed radix digits and moduli
             must be computed before the actual GCD can be recovered.
  
  Based on initial work by
  Authors:  Justin Brew, Anthony Rizzo, Kenneth Weber
            Mount Union College
            June 25, 2009

  Further revisions by 
  K. Weber  University of Mount Union
            weberk@mountunion.edu
            
  See GmpCudaDevice.cu for revision history.
*/

//  Enforce use of CUDA 9 or higher at compile time.
#if defined(CUDART_VERSION) && CUDART_VERSION >= 9000
#else
#error Requires CUDA 9 or more recent
#endif

#include <cassert>
#include <cstring>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include "GmpCuda.h"
//  Include the devicesRcpNoCheck definition, which is generated
//  by a configuration script.
#include "GmpCudaDevice-gcdDevicesRcpNoCheck.h"

//  Include the fastReciprocal and quasiQuoRem inline functions,
//  which are in a separate header file so that quasiQuoRem<false> can be certified
//  for use on specific devices.
#include "quasiQuoRem.h"

using namespace GmpCuda;

static constexpr int WARPS_PER_BLOCK   = GmpCudaDevice::GCD_BLOCK_SZ / WARP_SZ;  
static constexpr unsigned FULL_MASK    = 0xFFFFFFFF;           //  Used in sync functions.
static constexpr uint64_t MODULUS_MASK = uint64_t{0xFFFFFFFF}; //  Mask for modulus portion of pair.
static constexpr int32_t  MOD_INFINITY = INT32_MIN;            //  Larger than any modulur value

static constexpr int RCP_THRESHOLD_NORM_CLZ  = 32 - RCP_THRESHOLD_EXPT;  //  # leading zeros in a normalized denominator.
static constexpr uint32_t RCP_THRESHOLD = 1 << RCP_THRESHOLD_EXPT;

//  Make the cuda architecture number available as a constexpr for all compilation phases.
static constexpr int CUDA_ARCH =
#ifdef __CUDA_ARCH__
  __CUDA_ARCH__
#else
  -1
#endif
  ;
  
typedef GmpCudaDevice::pair_t pair_t;  //  Used to pass back result.

//  This type is used to conveniently manipulate the modulus and its inverse.
typedef struct {uint32_t modulus; uint64_t inverse;} modulus_t;


//  Which thread in the warp satisfying the predicate has a nonzero value?
//  Uses ballot so that every multiprocessor (deterministically) chooses the same pair.
//  In case there is no winner, use the 0 from warpLane 0.
__device__
static
inline
int
findAnyNonZero(pair_t pair, bool predicate = true)
{
  return max(0, __ffs(__ballot_sync(FULL_MASK, predicate && pair.value != 0)) - 1);
}

//  Posts to the barrier one of the pair parameters whose value is not 0.
//  If no such value is found, a pair with a 0 value is posted.
//  Preconditions:  all threads in block participate.
__device__
static
void
postAnyPairPriorityNonzero(pair_t pair, GmpCudaBarrier &bar)
{
   __shared__ pair_t sharedPair[WARP_SZ];
   
  __syncthreads();  // protect shared memory against last call to this function.

  if (findAnyNonZero(pair) == threadIdx.x % WARP_SZ)
    sharedPair[threadIdx.x / WARP_SZ] = pair;

  __syncthreads();
  
  pair = sharedPair[findAnyNonZero(sharedPair[threadIdx.x], threadIdx.x < WARPS_PER_BLOCK)];
  
  bar.post(*reinterpret_cast<uint64_t *>(&pair));
}

//  Chooses one of the pairs in the barrier that doesn't have a 0 value;
//  chosen pair is returned in pair as result.
//  If there are no nonzero values, a pair with value 0 is returned.
//  Preconditions:  all threads in block participate.
//  Postcondition: every thread will have the same pair.
__device__
static
void
collectAnyPairPriorityNonzero(pair_t& __restrict__ pair, 
                              GmpCudaBarrier& __restrict__ bar)
{
  __shared__ pair_t sharedPair[WARP_SZ];
  
  bar.collect(*reinterpret_cast<uint64_t*>(&pair)); // Only low gridDim.x threads have "good" values.
  
  __syncthreads();  // protect shared memory against last call to this function.
  
  int warpLane = threadIdx.x % WARP_SZ;
  
  if (findAnyNonZero(pair, threadIdx.x < gridDim.x) == warpLane && threadIdx.x < gridDim.x)
    sharedPair[threadIdx.x / WARP_SZ] = pair;

  __syncthreads();

  int numWarps = (gridDim.x - 1) / WARP_SZ + 1;

  //  All warps do this and get common value for winner.
  pair = sharedPair[findAnyNonZero(sharedPair[warpLane], warpLane < numWarps)];
}

//  Calculate min of x into lane 0 of warp.
__device__
inline
void
minWarp(uint64_t &x)
{
#pragma unroll
  for (int i = WARP_SZ/2; i > 0; i /= 2)
    x = min(x, __shfl_down_sync(FULL_MASK, x, i));
}

//  Calculates abs(x), except that MOD_INFINITY == INT32_MIN is not changed.
__device__
static
inline
uint64_t
modAbs(int32_t x)
{
  return (x < 0) ? ~x + 1 : x;
}

//  Posts pair which achieves the minimum of the absolute value 
//  of all pairs in each threadblock to bar.
//  Precondition: modulus of each pair is odd and all threads participate.
//  Postcondition: bar is ready for collectMinPair to be called.
__device__
static
void
postMinPair(pair_t pair, GmpCudaBarrier& bar)
{
  __shared__ uint64_t sharedX[WARP_SZ];

  __syncthreads();  // protect shared memory against last call to this function.
    
  //  Prepare a long int composed of the absolute value of pair.value in the high bits and pair.modulus in the low bits.
  //  Store sign of pair.value in the low bit of pair.modulus, which should always be 1 since it's odd.
  uint64_t x = (modAbs(pair.value) << 32) | (pair.modulus - (pair.value >= 0)); 

  //  Find the smallest in each warp, and store in sharedX.
  minWarp(x);
  if (threadIdx.x % WARP_SZ == 0)
    sharedX[threadIdx.x / WARP_SZ] = x;
  __syncthreads();

  //  Now find the min of the values in sharedX.
  //  WARPS_PER_BLOCK must be a power of 2 <= WARP_SZ.
  if (threadIdx.x < WARP_SZ)
    {
      x = sharedX[threadIdx.x];
#pragma unroll
      for (int i = WARPS_PER_BLOCK/2; i > 0; i /= 2)
        x = min(x, __shfl_down_sync(FULL_MASK, x, i));        
    }

  bar.post(x);
}

//  Returns, in pair, the pair which achieves the global minimum of the absolute value 
//  of the value over all the pairs that have been posted to bar.
//  Precondition: postMinPair was previously called and all threads participate.
__device__
static
void
collectMinPair(pair_t& __restrict__ pair, GmpCudaBarrier& __restrict__ bar)
{
  uint64_t x;
  bar.collect(x);
  
  __shared__ uint64_t sharedX[WARP_SZ];
  
  __syncthreads();  // protect shared memory against last call to this function.
      
  int numWarps =  (gridDim.x - 1) / WARP_SZ + 1;

  if (threadIdx.x / WARP_SZ < numWarps)
    {
      if (threadIdx.x >= gridDim.x)
        x = UINT64_MAX;
      minWarp(x);
      if (threadIdx.x % WARP_SZ == 0)
        sharedX[threadIdx.x / WARP_SZ] = x;
    }

  __syncthreads();
  if (threadIdx.x < WARP_SZ)
    {
      x = (threadIdx.x < numWarps) ? sharedX[threadIdx.x] : UINT64_MAX;
#pragma unroll
      for (int i = WARPS_PER_BLOCK/2; i > 1; i /= 2)  //  assert(gridDim.x <= blockDim.x);
        x = min(x, __shfl_down_sync(FULL_MASK, x, i));  
      sharedX[threadIdx.x] = min(x, __shfl_down_sync(FULL_MASK, x, 1));                            
   }

  __syncthreads();
  x = sharedX[0];
  
  pair.modulus = static_cast<uint32_t>(x & MODULUS_MASK); 
  pair.value   = static_cast<int32_t>(x >> 32);
  //  Restore original sign.
  if (pair.modulus & 1)
    pair.value = ~pair.value + 1;  // Should leave MOD_INFINITY unchanged.
  pair.modulus |= 1;
}

//  Determines whether the modulus is equal to x.
__device__
static
inline
bool
equals(uint32_t x, modulus_t m)
{
  return (m.modulus == x);
}

//  Return a - b (mod m) in the range 0..m-1.
//  Precondition: a, b are both in the range 0..m-1.
__device__
static
inline
uint32_t
modSub(uint32_t a, uint32_t b, modulus_t m)
{
  return a - b + (-(a < b) & m.modulus);
}

//  Calculate x mod m, where x is 64 bits long.
__device__
static
inline
uint32_t
mod(uint64_t x, modulus_t m)
{
  return x - static_cast<uint64_t>(m.modulus) * (__umul64hi(m.inverse, x) >> (L - 1));
}

//  Return a * b (mod m) in the range 0..m-1.
//  Precondition: a, b are both in the range 0..m-1, and m is prime.
__device__
static
inline
uint32_t
modMul(uint32_t a, uint32_t b, modulus_t m)
{
  return mod(static_cast<uint64_t>(a) * b, m);
}

__device__
static
inline
uint32_t
fromSigned(int32_t x, modulus_t m)
{
  return (x < 0) ? x + m.modulus : x;
}

// Give x mod m as a signed value in the range [-modulus/2, modulus,2]
__device__
static
inline
int32_t
toSigned(uint32_t x, modulus_t m)
{
  return (x >= m.modulus/2) ? x - m.modulus : x;
}

template
<typename T>
__device__
static
inline
void
swap(T& __restrict__ x, T& __restrict__ y)
{
  T tmp = x;
  x = y;
  y = tmp;
}


//  Computes an approximation for x / y, when x, y >= 2^21.
//  Approximation could be too small by 1 or 2.
//  The estimate of q from multiplying by the reciprocal here could be too high or too low by 1;
//  make it too low by 1 or 2, by subtracting 1.0 BEFORE truncating toward zero.
__device__
static
inline
uint32_t
quasiQuo(uint32_t x, uint32_t y)
{ 
  return __float2uint_rz(__fmaf_rz(__uint2float_rz(x), fastReciprocal(__uint2float_rz(y)), -1.0f));
}

//  Assumes x >= RCP_THRESHOLD > y. (Recall that RCP_THRESHOLD == 2^RCP_THRESHOLD_EXPT.)
//  First computes i such that 2^RCP_THRESHOLD_EXPT > y * 2^i >= 2^(RCP_THRESHOLD_EXPT - 1.
//  Returns q = 2^i * q' such that x - q' * y * 2^i < 2 * y * 2^i,
//  i.e., x - q * y < y * 2^(i + 1) < 2*RCP_THRESHOLD.
__device__
static
inline
uint32_t
quasiQuoNorm(uint32_t x, uint32_t y)
{
  int i = __clz(y) - RCP_THRESHOLD_NORM_CLZ;
  return quasiQuo(x, y << i) << i;
}

  
//  Faster divide possible when x and y are close in size.
//  Precondition: 2^32 > x, y >= 2^RCP_THRESHOLD_EXPT, so 0 <= x / y < 2^RCP_THRESHOLD_NORM_CLZ.
//  Could produce a quotient that's too small by 1--but modInv can tolerate that.
__device__
static
inline
uint32_t
quasiQuoRem(uint32_t& x, uint32_t y)
{ 
//  Computes an approximation q for x / y, when x, y >= RCP_THRESHOLD.
//  q could be too small by 1 or 2.
//  The estimate of q from multiplying by the reciprocal here could be too high or too low by 1;
//  make it too low by 1 or 2, by subtracting 1.0 BEFORE truncating toward zero.
//  uint32_t q = __float2uint_rz(__fmaf_rz(__uint2float_rz(x), fastReciprocal(__uint2float_rz(y)), -1.0f));
  uint32_t q = quasiQuo(x, y);
  x -= q * y; 
  if (x >= y)  //  Now x < 3 * y.
    x -= y, q += 1;
  return q;               //  Now x < 2 * y, but unlikely that x >= y.
}

//  Return 1/v (mod u), assuming gcd(u,v) == 1.
//  Assumes u > v > 0.
//  Uses the extended Euclidean algorithm:
//  see Knuth, The Art of Computer Programming, vol. 2, 3/e,
//  Algorithm X on pp342-3.
template <bool CHECK_RCP>
__device__
static
uint32_t
modInv(uint32_t u, uint32_t v)
{
  constexpr bool QUASI_TRANSITION = (CUDA_ARCH != 700);
  
  uint32_t u2 = 0, u3 = u;
  uint32_t v2 = 1, v3 = v;
  
  //  When u3 and v3 are both large enough, divide with floating point hardware.
  while  (v3 >= RCP_THRESHOLD)
    {
      u2 += v2 * quasiQuoRem(u3, v3);
      if (u3 <  RCP_THRESHOLD)
        break;
      v2 += u2 * quasiQuoRem(v3, u3);
    }
    
  bool negateResult = (v3 > u3);
  if  (negateResult)
    {
      swap(u2, v2);
      swap(u3, v3);
    }

  //  u3 >= RCP_THRESHOLD > v3.
  //  Transition to both u3 and v3 small, so values are cast into floats.
  //  Although algorithm can tolerate a quasi-quotient here (i.e., possibly one less than
  //  the true quotient), the true quotient is about as fast as the quasi-quotient,
  //  so we decide which version to use when the compiler compiles to a specific architecture.
  uint32_t q = (QUASI_TRANSITION) ? quasiQuoNorm(u3, v3) : u3 / v3;
  float u3f = __uint2float_rz(u3 - q * v3);
  float v3f = __uint2float_rz(v3);
  if (QUASI_TRANSITION)
    q += quasiQuoRem<CHECK_RCP>(u3f, v3f);
  u2 += v2 * q;
   
  //  When u3 and v3 are both small enough, divide with floating point hardware.   
  //  At this point v3f > u3f.
  //  The loop will stop when u3f <= 1.0.
  //  If u3f == 1.0, result is in u2.
  //  If u3f == 0.0, then v3f == 1.0 and result is in v2.
  //  If u3f ==-1.0, result is in u2.
  while (u3f > 1.0f)
    {
      v2 += u2 * quasiQuoRem<CHECK_RCP>(v3f, u3f);
      u2 += v2 * quasiQuoRem<CHECK_RCP>(u3f, v3f);
    }
  
  //  If we are transitioning with a quasi quotient and don't check the reciprocal, 
  //  u3f == -1.0f is possible, in which case the result will need to be negated.
  if (QUASI_TRANSITION && !CHECK_RCP)
    negateResult ^= (u3f == -1.0f);
    
  negateResult ^= (v3f != 1.0f);  //  Update negateResult based on where the answer ended up.
  
  if (v3f != 1.0f)                //  Answer in u2--copy into v2.
    v2 = u2;
  if (negateResult)
    v2 = u - v2;
  return v2;
}

// Calculate u/v mod m, in the range [0,m-1]
template <bool CHECK_RCP>
__device__
static
inline
uint32_t
modDiv(uint32_t u, uint32_t v, modulus_t m)
{
  return modMul(u, modInv<CHECK_RCP>(m.modulus, v), m);
}

//  Calculate x mod m for a multiword unsigned integer x.
__device__
static
uint32_t
modMP(uint32_t x[], size_t xSz, modulus_t m)
{
  __shared__ uint32_t sharedX[WARP_SZ];
  uint64_t result = uint64_t{0};
  
  __syncthreads();  // protect shared memory against last call to this function.
  
  while (xSz > warpSize)
    {
      xSz -= warpSize;
      //  Copy a block of x to shared memory for processing.
      if (threadIdx.x < warpSize)
        sharedX[threadIdx.x] = x[threadIdx.x + xSz];
      __syncthreads();
      //  Process the block in shared memory.
      for (size_t i = warpSize; i-- != 0;  )
        result = mod(result << 32 | sharedX[i], m);
      __syncthreads();
    }
  //  Now xSz <= warpSize.  Copy remainder of x to shared memory and process.
  if (threadIdx.x < xSz)
    sharedX[threadIdx.x] = x[threadIdx.x];
  __syncthreads();
  for (size_t i = xSz; i-- != 0;  )
    result = mod(result << 32 | sharedX[i], m);
  return static_cast<uint32_t>(result);
}
 
// Initialize modulus for this thread by reading a modulus m from the list
// and computing its "inverse", mInverse == 2^(W + L - 1) / m + 1.
__device__
static
inline
modulus_t
getModulus(uint32_t* moduliList)
{
    uint32_t m = moduliList[blockDim.x * blockIdx.x + threadIdx.x];
    uint64_t D = static_cast<uint64_t>(m);
    constexpr uint64_t FC_hi = uint64_t{1} << (W - 1);
    uint64_t q = FC_hi / D;
    uint64_t r = FC_hi % D;
    return {m, uint64_t{1} + (q << L) + (r << L) / D};
}

//  Device kernel for the GmpCudaDevice::getGcdKernel method.
template <bool CHECK_RCP>
__global__
static
void
kernel(uint32_t* __restrict__ buf, size_t uSz, size_t vSz, 
       uint32_t* __restrict__ moduliList, GmpCudaBarrier bar)
{
  int totalModuliRemaining = blockDim.x * gridDim.x;
  int ubits = (uSz + 1) * 32;  // somewhat of an overestimate
  int vbits = (vSz + 1) * 32;  // same here
  
  //MGCD1: [Find suitable moduli]
  modulus_t q = getModulus(moduliList);

  //MGCD2: [Convert to modular representation]

  uint32_t uq, vq;
  uq = modMP(buf,       uSz, q);
  vq = modMP(buf + uSz, vSz, q);

  //MGCD3: [reduction loop]

  bool active = true;  //  Is the modulus owned by this thread active, or has it been retired?

  pair_t pair, myPair;
  myPair.modulus = q.modulus;
  myPair.value = (vq == 0) ? MOD_INFINITY : toSigned(modDiv<CHECK_RCP>(uq, vq, q), q);
  postMinPair(myPair, bar);
  collectMinPair(pair, bar);
  
  do
    {
      uint32_t p, tq;
      int tbits;
      if (equals(pair.modulus, q))  //  Deactivate this modulus.
        active = false, myPair.value = MOD_INFINITY;
      if (active)
        {
          p = pair.modulus;
          if (p > q.modulus)        //  Bring within range.
            p -= q.modulus;
          tq = modDiv<CHECK_RCP>(modSub(uq, modMul(fromSigned(pair.value, q), vq, q), q), p, q);
          myPair.value = (tq == 0) ? MOD_INFINITY : toSigned(modDiv<CHECK_RCP>(vq, tq, q), q);
        }
      postMinPair(myPair, bar);
      if (active)
        uq = vq, vq = tq;       
      totalModuliRemaining -= 1;
      tbits = ubits - (L - 1) + __ffs(abs(pair.value));
      ubits = vbits, vbits = tbits;
      if (totalModuliRemaining * (L - 2) <= ubits)  //  Ran out of moduli--means initial estimate was wrong.
        {
          if (blockIdx.x && threadIdx.x)
            return;
          buf[0] = GmpCudaDevice::GCD_KERNEL_ERROR, buf[1] = GmpCudaDevice::GCD_REDUX_ERROR;
          return;
        }        
      collectMinPair(pair, bar);
    }
  while (pair.value != MOD_INFINITY);
   
  //MGCD4: [Find SIGNED mixed-radix representation] Each "digit" is either positive or negative.

  pair_t* pairs = (pair_t *)buf + 1;

  myPair.value = (active) ? toSigned(uq, q) : 0;  //  Inactive threads should have low priority.

  postAnyPairPriorityNonzero(myPair, bar);

  collectAnyPairPriorityNonzero(pair, bar);

  do
    {
      if (equals(pair.modulus, q))  //  deactivate modulus.
        active = false, myPair.value = 0;
      if (active)
        {
          uint32_t p = pair.modulus;
          if (pair.modulus > q.modulus)  //  Bring within range.
            p -= q.modulus;
          uq = modDiv<CHECK_RCP>(modSub(uq, fromSigned(pair.value, q), q), p, q);
          myPair.value = toSigned(uq, q);
        }
      postAnyPairPriorityNonzero(myPair, bar);
      *pairs++ = pair;
      totalModuliRemaining -= 1;
      if (totalModuliRemaining <= 0)  //  Something went wrong.
        break;
      collectAnyPairPriorityNonzero(pair, bar);
    }
  while (pair.value != 0);

  if (blockIdx.x | threadIdx.x)  //  Final cleanup by just one thread.
    return;

  //  Return a count of all the nonzero pairs, plus one more "pair" that includes buf[0] itself.
  //  If there aren't enough moduli to recover the result, return error codes.
  if (pair.value != 0) 
    buf[0] = GmpCudaDevice::GCD_KERNEL_ERROR, buf[1] = GmpCudaDevice::GCD_RECOVERY_ERROR;
  else
    buf[0] = pairs - reinterpret_cast<pair_t*>(buf);   
}

//  Used to find a device name in devicesRcpNoCheck.
static 
int 
comparator(const void* s1, const void* s2Ptr)
{
  return strcmp(static_cast<const char*>(s1), *static_cast<char * const *>(s2Ptr));
}

//  Return the appropriate gcd kernel for a device to use, based on
//  whether there needs to be a check performed after the rcp.approx reciprocal
//  in quasiQuoRem.
const 
void* 
GmpCudaDevice::getGcdKernel(char* devName)
{
  void* key = bsearch(static_cast<const void*>(devName), static_cast<const void*>(devicesRcpNoCheck), 
                      sizeof(devicesRcpNoCheck)/sizeof(char*), sizeof(char*), &comparator);
  return reinterpret_cast<const void *>((key == NULL) ? &kernel<true> : &kernel<false>);
}

