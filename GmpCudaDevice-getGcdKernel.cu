#include "hip/hip_runtime.h"
/*  GmpCudaDevice-getGcdKernel.cu -- provides GmpCudaDevice::getGcdKernel method
                                     (includes the gcd kernel code).

  Implementation of the modular integer gcd algorithm using L <= 32 bit moduli.
  
  Reference: Weber, Trevisan, Martins 2005. A Modular Integer GCD algorithm
             Journal of Algorithms 54, 2 (February, 2005) 152-167.

             Note that there is an error in Fig. 2, which shows that the
             final result can be recovered as the mixed radix representation
             is calculated.  In actuality, all the mixed radix digits and moduli
             must be computed before the actual GCD can be recovered.
  
  Based on initial work by
  Authors:  Justin Brew, Anthony Rizzo, Kenneth Weber
            Mount Union College
            June 25, 2009

  Further revisions by 
  K. Weber  University of Mount Union
            weberk@mountunion.edu
            
  See GmpCudaDevice.cu for revision history.
*/

//  Enforce use of CUDA 9 or higher at compile time.
#if defined(CUDART_VERSION) && CUDART_VERSION >= 9000
#else
#error Requires CUDA 9 or more recent
#endif

#include <cassert>
#include <cstring>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include "GmpCuda.h"
#include "GmpCudaDevice-gcdDevicesQuoRemQuasi.h"
#include "modInv.h"

using namespace GmpCuda;

static constexpr int      WARPS_PER_BLOCK = GmpCudaDevice::GCD_BLOCK_SZ / WARP_SZ;

static constexpr unsigned FULL_MASK    = 0xFFFFFFFF;           //  Used in sync functions.
static constexpr uint64_t MODULUS_MASK = uint64_t{0xFFFFFFFF}; //  Mask for modulus portion of pair.
static constexpr int32_t  MOD_INFINITY = INT32_MIN;            //  Larger than any modulur value

typedef GmpCudaDevice::pair_t pair_t;  //  Used to pass back result.

//  This type is used to conveniently manipulate the modulus and its inverse.
typedef struct {uint32_t modulus; uint64_t inverse;} modulus_t;


//  Which thread in the warp satisfying the predicate has a nonzero value?
//  Uses ballot so that every multiprocessor (deterministically) chooses the same pair.
//  In case there is no winner, use the 0 from warpLane 0.
__device__
static
inline
int
findAnyNonZero(pair_t pair, bool predicate = true)
{
  return max(0, __ffs(__ballot_sync(FULL_MASK, predicate && pair.value != 0)) - 1);
}

//  Posts to the barrier one of the pair parameters whose value is not 0.
//  If no such value is found, a pair with a 0 value is posted.
//  Preconditions:  all threads in block participate.
__device__
static
void
postAnyPairPriorityNonzero(pair_t pair, GmpCudaBarrier &bar, int devIdx, int devDim)
{
   __shared__ pair_t sharedPair[WARP_SZ];
   
  __syncthreads();  // protect shared memory against last call to this function.

  if (findAnyNonZero(pair) == threadIdx.x % WARP_SZ)
    sharedPair[threadIdx.x / WARP_SZ] = pair;

  __syncthreads();
  
  pair = sharedPair[findAnyNonZero(sharedPair[threadIdx.x], threadIdx.x < WARPS_PER_BLOCK)];
  
  bar.post(*reinterpret_cast<uint64_t *>(&pair), devIdx, devDim);
}

//  Chooses one of the pairs in the barrier that doesn't have a 0 value;
//  chosen pair is returned in pair as result.
//  If there are no nonzero values, a pair with value 0 is returned.
//  Preconditions:  all threads in block participate.
//  Postcondition: every thread will have the same pair.
__device__
static
void
collectAnyPairPriorityNonzero(pair_t& __restrict__ pair, 
                              GmpCudaBarrier& __restrict__ bar, int devIdx, int devDim)
{
  __shared__ pair_t sharedPair[WARP_SZ];
  
  bar.collect(*reinterpret_cast<uint64_t*>(&pair), devIdx, devDim); // Only low devDim * gridDim.x threads have "good" values.
  
  __syncthreads();  // protect shared memory against last call to this function.
  
  int warpLane = threadIdx.x % WARP_SZ;
  
  if (findAnyNonZero(pair, threadIdx.x < devDim * gridDim.x) == warpLane && threadIdx.x < devDim * gridDim.x)
    sharedPair[threadIdx.x / WARP_SZ] = pair;

  __syncthreads();

  int numWarps = (devDim * gridDim.x - 1) / WARP_SZ + 1;

  //  All warps do this and get common value for winner.
  pair = sharedPair[findAnyNonZero(sharedPair[warpLane], warpLane < numWarps)];
}

//  Calculate min of x into lane 0 of warp.
__device__
inline
void
minWarp(uint64_t &x)
{
#pragma unroll
  for (int i = WARP_SZ/2; i > 0; i /= 2)
    x = min(x, __shfl_down_sync(FULL_MASK, x, i));
}

//  Calculates abs(x), except that MOD_INFINITY == INT32_MIN is not changed.
__device__
static
inline
uint64_t
modAbs(int32_t x)
{
  return (x < 0) ? ~x + 1 : x;
}

//  Posts pair which achieves the minimum of the absolute value 
//  of all pairs in each threadblock to bar.
//  Precondition: modulus of each pair is odd and all threads participate.
//  Postcondition: bar is ready for collectMinPair to be called.
__device__
static
void
postMinPair(pair_t pair, GmpCudaBarrier& bar, int devIdx, int devDim)
{
  __shared__ uint64_t sharedX[WARP_SZ];

  __syncthreads();  // protect shared memory against last call to this function.
    
  //  Prepare a long int composed of the absolute value of pair.value in the high bits and pair.modulus in the low bits.
  //  Store sign of pair.value in the low bit of pair.modulus, which should always be 1 since it's odd.
  uint64_t x = (modAbs(pair.value) << 32) | (pair.modulus - (pair.value >= 0)); 

  //  Find the smallest in each warp, and store in sharedX.
  minWarp(x);
  if (threadIdx.x % WARP_SZ == 0)
    sharedX[threadIdx.x / WARP_SZ] = x;
  __syncthreads();

  //  Now find the min of the values in sharedX.
  //  WARPS_PER_BLOCK must be a power of 2 <= WARP_SZ.
  if (threadIdx.x < WARP_SZ)
    {
      x = sharedX[threadIdx.x];
#pragma unroll
      for (int i = WARPS_PER_BLOCK/2; i > 0; i /= 2)
        x = min(x, __shfl_down_sync(FULL_MASK, x, i));        
    }

  bar.post(x, devIdx, devDim);
}

//  Returns, in pair, the pair which achieves the global minimum of the absolute value 
//  of the value over all the pairs that have been posted to bar.
//  Precondition: postMinPair was previously called and all threads participate.
__device__
static
void
collectMinPair(pair_t& __restrict__ pair, GmpCudaBarrier& __restrict__ bar, int devIdx, int devDim)
{
  uint64_t x;
  bar.collect(x, devIdx, devDim);
  
  __shared__ uint64_t sharedX[WARP_SZ];
  
  __syncthreads();  // protect shared memory against last call to this function.
      
  int numWarps =  (devDim * gridDim.x - 1) / WARP_SZ + 1;

  if (threadIdx.x / WARP_SZ < numWarps)
    {
      if (threadIdx.x >= devDim * gridDim.x)
        x = UINT64_MAX;
      minWarp(x);
      if (threadIdx.x % WARP_SZ == 0)
        sharedX[threadIdx.x / WARP_SZ] = x;
    }

  __syncthreads();
  if (threadIdx.x < WARP_SZ)
    {
      x = (threadIdx.x < numWarps) ? sharedX[threadIdx.x] : UINT64_MAX;
#pragma unroll
      for (int i = WARPS_PER_BLOCK/2; i > 1; i /= 2)  //  assert(devDim * gridDim.x <= blockDim.x);
        x = min(x, __shfl_down_sync(FULL_MASK, x, i));  
      sharedX[threadIdx.x] = min(x, __shfl_down_sync(FULL_MASK, x, 1));                            
   }

  __syncthreads();
  x = sharedX[0];
  
  pair.modulus = static_cast<uint32_t>(x & MODULUS_MASK); 
  pair.value   = static_cast<int32_t>(x >> 32);
  //  Restore original sign.
  if (pair.modulus & 1)
    pair.value = ~pair.value + 1;  // Should leave MOD_INFINITY unchanged.
  pair.modulus |= 1;
}

//  Determines whether the modulus is equal to x.
__device__
static
inline
bool
equals(uint32_t x, modulus_t m)
{
  return (m.modulus == x);
}

//  Return a - b (mod m) in the range 0..m-1.
//  Precondition: a, b are both in the range 0..m-1.
__device__
static
inline
uint32_t
modSub(uint32_t a, uint32_t b, modulus_t m)
{
  return a - b + (-(a < b) & m.modulus);
}

//  Calculate x mod m, where x is 64 bits long.
__device__
static
inline
uint32_t
mod(uint64_t x, modulus_t m)
{
  return x - static_cast<uint64_t>(m.modulus) * (__umul64hi(m.inverse, x) >> (L - 1));
}

//  Return a * b (mod m) in the range 0..m-1.
//  Precondition: a, b are both in the range 0..m-1, and m is prime.
__device__
static
inline
uint32_t
modMul(uint32_t a, uint32_t b, modulus_t m)
{
  return mod(static_cast<uint64_t>(a) * b, m);
}

__device__
static
inline
uint32_t
fromSigned(int32_t x, modulus_t m)
{
  return (x < 0) ? x + m.modulus : x;
}

// Give x mod m as a signed value in the range [-modulus/2, modulus,2]
__device__
static
inline
int32_t
toSigned(uint32_t x, modulus_t m)
{
  return (x >= m.modulus/2) ? x - m.modulus : x;
}

// Calculate u/v mod m, in the range [0,m-1]
template <QuoRemType QRTYPE>
__device__
static
inline
uint32_t
modDiv(uint32_t u, uint32_t v, modulus_t m)
{
  return modMul(u, modInv<QRTYPE>(v, m.modulus), m);
}

//  Calculate x mod m for a multiword unsigned integer x.
__device__
static
uint32_t
modMP(uint32_t x[], size_t xSz, modulus_t m)
{
  __shared__ uint32_t sharedX[WARP_SZ];
  uint64_t result = uint64_t{0};
  
  __syncthreads();  // protect shared memory against last call to this function.
  
  while (xSz > warpSize)
    {
      xSz -= warpSize;
      //  Copy a block of x to shared memory for processing.
      if (threadIdx.x < warpSize)
        sharedX[threadIdx.x] = x[threadIdx.x + xSz];
      __syncthreads();
      //  Process the block in shared memory.
      for (size_t i = warpSize; i-- != 0;  )
        result = mod(result << 32 | sharedX[i], m);
      __syncthreads();
    }
  //  Now xSz <= warpSize.  Copy remainder of x to shared memory and process.
  if (threadIdx.x < xSz)
    sharedX[threadIdx.x] = x[threadIdx.x];
  __syncthreads();
  for (size_t i = xSz; i-- != 0;  )
    result = mod(result << 32 | sharedX[i], m);
  return static_cast<uint32_t>(result);
}
 
// Initialize modulus for this thread by reading a modulus m from the list
// and computing its "inverse", mInverse == 2^(W + L - 1) / m + 1.
__device__
static
inline
modulus_t
getModulus(uint32_t* moduliList, int devIdx)
{
    uint32_t m = moduliList[gridDim.x * blockDim.x * devIdx + blockDim.x * blockIdx.x + threadIdx.x];
    uint64_t D = static_cast<uint64_t>(m);
    constexpr uint64_t FC_hi = uint64_t{1} << (W - 1);
    uint64_t q = FC_hi / D;
    uint64_t r = FC_hi % D;
    return {m, uint64_t{1} + (q << L) + (r << L) / D};
}

//  Device kernel for the GmpCudaDevice::getGcdKernel method.
template <QuoRemType QRTYPE>
__global__
static
void
kernel(uint32_t* __restrict__ buf, size_t uSz, size_t vSz, 
       uint32_t* __restrict__ moduliList, GmpCudaBarrier bar,
       int devIdx, int devDim)
{

  if (devIdx == 0)
    {
      if (blockIdx.x == 0 && threadIdx.x == 0)
        printf("From device %d of %d\n",  devIdx, devDim);
    }
  else
    {
      if (blockIdx.x == 0 && threadIdx.x == 0)
        printf("From device %d of %d\n", devIdx, devDim);
      return;
    }
  int totalModuliRemaining = blockDim.x * gridDim.x * devDim;
  int ubits = (uSz + 1) * 32;  // somewhat of an overestimate
  int vbits = (vSz + 1) * 32;  // same here

  //MGCD1: [Find suitable moduli]
  modulus_t q = getModulus(moduliList, devIdx);

  //MGCD2: [Convert to modular representation]

  uint32_t uq, vq;
  uq = modMP(buf,       uSz, q);
  vq = modMP(buf + uSz, vSz, q);

  //MGCD3: [reduction loop]

  bool active = true;  //  Is the modulus owned by this thread active, or has it been retired?

  pair_t pair, myPair;
  myPair.modulus = q.modulus;
  myPair.value = (vq == 0) ? MOD_INFINITY : toSigned(modDiv<QRTYPE>(uq, vq, q), q);
  postMinPair(myPair, bar, devIdx, devDim);
  collectMinPair(pair, bar, devIdx, devDim);
  
  do
    {
      uint32_t p, tq;
      int tbits;
      if (equals(pair.modulus, q))  //  Deactivate this modulus.
        active = false, myPair.value = MOD_INFINITY;
      if (active)
        {
          p = pair.modulus;
          if (p > q.modulus)        //  Bring within range.
            p -= q.modulus;
          tq = modDiv<QRTYPE>(modSub(uq, modMul(fromSigned(pair.value, q), vq, q), q), p, q);
          myPair.value = (tq == 0) ? MOD_INFINITY : toSigned(modDiv<QRTYPE>(vq, tq, q), q);
        }
      postMinPair(myPair, bar, devIdx, devDim);
      if (active)
        uq = vq, vq = tq;       
      totalModuliRemaining -= 1;
      tbits = ubits - (L - 1) + __ffs(abs(pair.value));
      ubits = vbits, vbits = tbits;
      if (totalModuliRemaining * (L - 2) <= ubits)  //  Ran out of moduli--means initial estimate was wrong.
        {
          if (devIdx | blockIdx.x | threadIdx.x)
            return;
          buf[0] = GmpCudaDevice::GCD_KERNEL_ERROR, buf[1] = GmpCudaDevice::GCD_REDUX_ERROR;
          return;
        }        
      collectMinPair(pair, bar, devIdx, devDim);
    }
  while (pair.value != MOD_INFINITY);
   
  //MGCD4: [Find SIGNED mixed-radix representation] Each "digit" is either positive or negative.

  pair_t* pairs = (pair_t *)buf + 1;

  myPair.value = (active) ? toSigned(uq, q) : 0;  //  Inactive threads should have low priority.

  postAnyPairPriorityNonzero(myPair, bar, devIdx, devDim);

  collectAnyPairPriorityNonzero(pair, bar, devIdx, devDim);

  do
    {
      if (equals(pair.modulus, q))  //  deactivate modulus.
        active = false, myPair.value = 0;
      if (active)
        {
          uint32_t p = pair.modulus;
          if (pair.modulus > q.modulus)  //  Bring within range.
            p -= q.modulus;
          uq = modDiv<QRTYPE>(modSub(uq, fromSigned(pair.value, q), q), p, q);
          myPair.value = toSigned(uq, q);
        }
      postAnyPairPriorityNonzero(myPair, bar, devIdx, devDim);
      *pairs++ = pair;
      totalModuliRemaining -= 1;
      if (totalModuliRemaining <= 0)  //  Something went wrong.
        break;
      collectAnyPairPriorityNonzero(pair, bar, devIdx, devDim);
    }
  while (pair.value != 0);

  if (devIdx | blockIdx.x | threadIdx.x)  //  Final cleanup by just one thread.
    return;

  //  Return a count of all the nonzero pairs, plus one more "pair" that includes buf[0] itself.
  //  If there aren't enough moduli to recover the result, return error codes.
  if (pair.value != 0) 
    buf[0] = GmpCudaDevice::GCD_KERNEL_ERROR, buf[1] = GmpCudaDevice::GCD_RECOVERY_ERROR;
  else
    buf[0] = pairs - reinterpret_cast<pair_t*>(buf);   
}

__global__
static
void
checkFastReciprocal(bool* pass)
{
  *pass = (fastReciprocal(1.0f) == 1.0f && fastReciprocal(2.0f) == 0.5f);
}

//  Return the appropriate gcd kernel for a device to use, based on
//  whether the device supports quoRem<QUASI>, quoRem<FAST_EXACT>, or quoRem<SAFE_EXACT>.
const 
void* 
GmpCudaDevice::getGcdKernel(char* devName)
{
  void* ptr = bsearch(static_cast<const void*>(devName), 
                      static_cast<const void*>(devicesQuoRemQuasi), 
                      sizeof(devicesQuoRemQuasi)/sizeof(char*), 
                      sizeof(char*),
                      [](const void* s1, const void* s2Ptr) -> int
                        {
                          return strcmp(static_cast<const char*>(s1), *static_cast<char * const *>(s2Ptr));
                        }
                     );
  if (ptr != NULL)
    return reinterpret_cast<const void *>(&kernel<QUASI>);
  bool* globalPass;
  bool pass;
  assert(hipSuccess == hipMalloc(&globalPass, sizeof(pass)));
  checkFastReciprocal<<<1,1>>>(globalPass);
  assert(hipSuccess == hipDeviceSynchronize());
  assert(hipSuccess == hipMemcpy(&pass, globalPass, sizeof(pass), hipMemcpyDeviceToHost));
  assert(hipSuccess == hipFree(globalPass));
  return reinterpret_cast<const void *>((pass) ? &kernel<FAST_EXACT> : &kernel<SAFE_EXACT>);
}

