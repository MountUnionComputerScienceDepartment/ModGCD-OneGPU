#include "hip/hip_runtime.h"
/*  certifyQuasiQuoRem.cu

    This program will certify that quasiQuoRem<false>(xf, yf) works correctly, 
    as long as xf and yf are integers and 1 <= xf, yf < 2^22,
    by testing the function on all possible input satisfying the preconditions.
    
    K. Weber
    20-July, 2018.
*/

#include <hip/hip_runtime.h>
#include <cassert>
#include <stdio.h>
#include <stdint.h>
#include "quasiQuoRem.h"

__global__ void kernel(bool* fail)
{
  constexpr uint32_t LIMIT = 1 << 22;

  for (uint32_t y = blockIdx.x * blockDim.x + threadIdx.x  + 1; y < LIMIT; y += blockDim.x * gridDim.x)
    {
      float yf = __uint2float_rz(y);
      for (uint32_t x = 1; x < LIMIT; x += 1)
        {
          float xf = __uint2float_rz(x);
          float qf = quasiQuoRem<false>(xf, yf);
          if (xf >= 0.0f)
            continue;
          *fail = true;
          printf("Failed for x == %u and y == %u: qf == %f, xf = %f\n", x, y, qf, xf);
          return;
        }
    }
}


int main(void)
{
  printf("Starting\n");
  fflush(0);
  bool fail = false;
  bool* globalFail;
  assert(hipSuccess == hipMalloc(&globalFail, sizeof(fail)));
  assert(hipSuccess == hipMemcpy(globalFail, &fail, sizeof(fail), hipMemcpyHostToDevice));
  struct hipDeviceProp_t props;
  assert(hipSuccess == hipGetDeviceProperties(&props, 0));
  kernel<<<props.multiProcessorCount,1024>>>(globalFail);
  assert(hipSuccess == hipDeviceSynchronize());
  assert(hipSuccess == hipMemcpy(&fail, globalFail, sizeof(fail), hipMemcpyDeviceToHost));
  printf("Device %s: %s.\n", props.name, fail ? "FAIL" : "PASS");
  printf("All done\n");
}
